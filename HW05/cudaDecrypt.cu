#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include "functions.c"

//compute a*b mod p safely
__device__ unsigned int modprodcu(unsigned int a, unsigned int b, unsigned int p) {
  unsigned int za = a;
  unsigned int ab = 0;

  while (b > 0) {
    if (b%2 == 1) ab = (ab +  za) % p;
    za = (2 * za) % p;
    b /= 2;
  }
  return ab;
}

//compute a^b mod p safely
__device__ unsigned int modExpcu(unsigned int a, unsigned int b, unsigned int p) {
  unsigned int z = a;
  unsigned int aExpb = 1;

  while (b > 0) {
    if (b%2 == 1) aExpb = modprodcu(aExpb, z, p);
    z = modprodcu(z, z, p);
    b /= 2;
  }
  return aExpb;
}
__global__ void search(int N, unsigned int p, unsigned int g, unsigned int h, unsigned int* x){

    unsigned int myX = (unsigned int)(threadIdx.x+blockIdx.x*blockDim.x);
	unsigned int myY = (unsigned int)(threadIdx.y+blockIdx.y*blockDim.y);

	//find the secret key
	unsigned int i = myY*N+myX;
	if(i < p) {
    	if (modExpcu(g,i+1,p)==h)
       		*x=i+1;
	}
}
int main (int argc, char **argv) {

  /* Part 2. Start this program by first copying the contents of the main function from 
     your completed decrypt.c main function. */

  //declare storage for an ElGamal cryptosytem
  unsigned int n, p, g, h;
  unsigned int Nints;

  //get the secret key from the user
  //printf("Enter the secret key (0 if unknown): "); fflush(stdout);
  //char stat = scanf("%u",&x);

  unsigned int* h_x;

  //printf("Reading file.\n");

  FILE* f = fopen("public_key.txt", "r");
  fscanf(f, "%u\n%u\n%u\n%u\n", &n, &p, &g, &h);
  fclose(f);
  f = fopen("message.txt", "r");
  fscanf(f, "%u\n", &Nints);
  unsigned int* Zmessage = (unsigned int*) malloc(Nints*sizeof(unsigned int));
  unsigned int* a = (unsigned int*) malloc(Nints*sizeof(unsigned int));
  for(int i = 0; i < Nints; i++){
	  fscanf(f, "%u %u\n", &Zmessage[i], &a[i]);
  }
  fclose(f);
//---------------------------------------------------------------------------------------------------------------
    
  unsigned int* d_x;
  hipMalloc(&d_x, sizeof(unsigned int));
  dim3 B(256, 256, 1);
  int N = (n-16)/2;
  if(N <= 0)
	  N = 1;
  else
	  N = 1 << N;
  dim3 G(N,N,1);

  double startTime = clock();
  search<<<G,B>>>(N, p, g, h, d_x);
  hipDeviceSynchronize();
  double endTime = clock();

  double totalTime = (endTime-startTime)/CLOCKS_PER_SEC;
  double work = (double) p;
  double throughput = work/totalTime;

  printf("Searching all keys took %g seconds, throughput was %g values tested per second.\n", totalTime, throughput);
  hipMemcpy(h_x,d_x,sizeof(unsigned int),hipMemcpyDeviceToHost);
  hipFree(d_x);
//--------------------------------------------------------------------------------------------------------------

  unsigned int Nchars = Nints*(n-1)/8;
  ElGamalDecrypt(Zmessage, a, Nints, p, *h_x);
  unsigned char* message = (unsigned char*) malloc(Nchars*sizeof(unsigned char));
  convertZToString(Zmessage, Nints, message, Nchars);
  printf("Decrypted message: \"%s\"\n", message);
  return 0;
  /* Q4 Make the search for the secret key parallel on the GPU using CUDA. */

  return 0;
}
