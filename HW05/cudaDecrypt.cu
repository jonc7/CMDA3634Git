#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <string.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include "functions.c"

//compute a*b mod p safely
__device__ unsigned int modprodcu(unsigned int a, unsigned int b, unsigned int p) {
  unsigned int za = a;
  unsigned int ab = 0;

  while (b > 0) {
    if (b%2 == 1) ab = (ab +  za) % p;
    za = (2 * za) % p;
    b /= 2;
  }
  return ab;
}

//compute a^b mod p safely
__device__ unsigned int modExpcu(unsigned int a, unsigned int b, unsigned int p) {
  unsigned int z = a;
  unsigned int aExpb = 1;

  while (b > 0) {
    if (b%2 == 1) aExpb = modprodcu(aExpb, z, p);
    z = modprodcu(z, z, p);
    b /= 2;
  }
  return aExpb;
}
__global__ void search(unsigned int p, unsigned int g, unsigned int h, unsigned int* x){

    unsigned int myX = (unsigned int)(threadIdx.x+blockIdx.x*blockDim.x);
	unsigned int myY = (unsigned int)(threadIdx.y+blockIdx.y*blockDim.y);

	//find the secret key
	unsigned int i = myY*blockDim.x*gridDim.x+myX;
	if(i < p) {
    	if (modExpcu(g,i+1,p)==h)
       		*x=i+1;
	}
}
int main (int argc, char **argv) {

  /* Part 2. Start this program by first copying the contents of the main function from 
     your completed decrypt.c main function. */

  //declare storage for an ElGamal cryptosytem
  unsigned int n, p, g, h;
  unsigned int Nints;

  //get the secret key from the user
  //printf("Enter the secret key (0 if unknown): "); fflush(stdout);
  //char stat = scanf("%u",&x);

  unsigned int* h_x = (unsigned int*)malloc(sizeof(unsigned int));
  *h_x = 0;

  //printf("Reading file.\n");

  FILE* f = fopen("bonus_public_key.txt", "r");
  fscanf(f, "%u\n%u\n%u\n%u\n", &n, &p, &g, &h);
  fclose(f);
  f = fopen("bonus_message.txt", "r");
  fscanf(f, "%u\n", &Nints);
  unsigned int* Zmessage = (unsigned int*) malloc(Nints*sizeof(unsigned int));
  unsigned int* a = (unsigned int*) malloc(Nints*sizeof(unsigned int));
  for(int i = 0; i < Nints; i++){
	  fscanf(f, "%u %u\n", &Zmessage[i], &a[i]);
  }
  fclose(f);
//---------------------------------------------------------------------------------------------------------------
    
  unsigned int* d_x;
  hipMalloc(&d_x, sizeof(unsigned int));
  dim3 B(32, 32, 1);
  int N = (n-10+1)/2;
  if(N < 0)
	  N = 0;
  N = 1 << N;
  dim3 G(N,N,1);

  double startTime = clock();
  search <<< G,B >>> (p, g, h, d_x);
  hipDeviceSynchronize();
  double endTime = clock();

  double totalTime = (endTime-startTime)/CLOCKS_PER_SEC;
  double work = (double) p;
  double throughput = work/totalTime;

  printf("Searching all keys took %g seconds, throughput was %g values tested per second.\n", totalTime, throughput);
  hipMemcpy(h_x,d_x,sizeof(unsigned int),hipMemcpyDeviceToHost);
  printf("x=%u\n", *h_x);
  hipFree(d_x);
//--------------------------------------------------------------------------------------------------------------

  unsigned int Nchars = Nints*(n-1)/8;
  printf("Nchars=%u\n", Nchars);
  ElGamalDecrypt(Zmessage, a, Nints, p, *h_x);
  unsigned char* message = (unsigned char*) malloc(Nchars*sizeof(unsigned char));
  convertZToString(Zmessage, Nints, message, Nchars);
  printf("Decrypted message: \"%s\"\n", message);
  free(h_x);
  return 0;
  /* Q4 Make the search for the secret key parallel on the GPU using CUDA. */
}
